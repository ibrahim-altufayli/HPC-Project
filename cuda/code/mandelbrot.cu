
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <hip/hip_complex.h>
#include <complex>
#include <chrono>
#include <string.h>
#include <cmath>

// Ranges of the set
#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1



#define DEGREE 2        // Degree of the polynomial

using namespace std;

__global__ void calcPxl(int* image, int width, int height, double step, int min_x, int min_y, int iterations){
    const int pos = threadIdx.x + blockIdx.x * blockDim.x;

        if (pos < width * height){

            image[pos] = 0;

            const int row = pos / width;
            const int col = pos % width;
            const complex<double> c(col * step + min_x, row * step + min_y);

            // z = z^2 + c
            complex<double> z(0, 0);
            for (int i = 1; i <= iterations; i++)
            {
                z = pow(z, 2) + c;

                // If it is convergent
                if (abs(z) >= 2)
                {
                    image[pos] = i;
                    break;
                }
            }
        }
    return;
    }


double calc_rmse(int * imageGen, string refImagePath){
    ifstream file(refImagePath.c_str()); 
    int row = 0;
    int width = 0;
    string line;
    string delimiter = ",";
    int sumOfDiff = 0;
    while (std::getline(file, line))
    {
        size_t delimiterPos = 0;
        int pxlCol = 0;
        while ((delimiterPos = line.find(delimiter)) != string::npos) {
            string pxlValueStr = line.substr(0, delimiterPos);
            char* p;
            int pxlValue = strtol(pxlValueStr.c_str(), &p, 10);
            line.erase(0, delimiterPos + delimiter.length());
            int pxl1DPos = row * width + pxlCol;
            sumOfDiff += pow(pxlValue - imageGen[pxl1DPos], 2);
            pxlCol +=1;
        }
        //handel last pixel
        string pxlValueStr = line;
        char* p;
        int pxlValue = strtol(pxlValueStr.c_str(), &p, 10);
        int pxl1DPos = row * width + pxlCol;
        sumOfDiff += pow(pxlValue - imageGen[pxl1DPos], 2);
        pxlCol +=1;
        if(width == 0){
            width = pxlCol;
        }
        row += 1;
    }
    int height = row;
    file.close();
    cout<<"width: "<<width<<" height: "<<height<<endl;
    return sqrt(sumOfDiff/(height * width));
}

int main(int argc, char **argv)
{
    int ITERATIONS = 1000, RESOLUTION = 1000;
    // Image ratio
    int RATIO_X = MAX_X - MIN_X;
    int RATIO_Y = MAX_Y - MIN_Y;


    if(argc > 1){
        char *p;
        RESOLUTION = strtol(argv[1], &p, 10);
        if (*p != '\0'){
            cout << "Please use only integer values for RESOLUTION" << endl;
            return -1;
        }
        if(argc > 2){
            p = NULL;
            ITERATIONS = strtol(argv[2], &p, 10);
            if(*p != '\0'){
                cout << "Please use only integer values for ITERATIONS" << endl;
                return -1;
            }

        }
    }
    cout<< "Image Resolution: "<< RESOLUTION<<endl;
    cout<< "#Iterations: "<< ITERATIONS <<endl;
    
    int nThreads = 4;
    if(argc > 3 && strlen(argv[3]) > 0){
        char* p;
        nThreads = strtol(argv[3], &p, 10);
        if (*p != '\0') {
            return 1; // In main(), returning non-zero means failure
        }
    }

    int WIDTH = RATIO_X * RESOLUTION;
    int HEIGHT = RATIO_Y * RESOLUTION;

    double STEP = ((double)RATIO_X / WIDTH);

    
    int N = WIDTH * HEIGHT;
    int * image = (int*)malloc( N * sizeof(int) );
    int *dev_image;

    const auto start = chrono::steady_clock::now();
    

    hipMalloc( (void**)&dev_image, N * sizeof(int) );
    hipMemcpy(dev_image, image, N * sizeof(int), hipMemcpyHostToDevice);
    dim3 threads(nThreads);
    dim3 blocks ( (N+threads.x-1)/threads.x );
    cout<<"Threads: "<<threads.x<<" Blocks: "<<blocks.x<<endl;

    calcPxl<<<blocks,threads>>>(dev_image, WIDTH, HEIGHT, STEP, MIN_X, MIN_Y, ITERATIONS);
    

    hipMemcpy(image, dev_image, N * sizeof(int), hipMemcpyDeviceToHost);


      const auto end = chrono::steady_clock::now();
    cout << "Time elapsed: "
         << chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << " milliseconds." << endl;

    
    
        ofstream results_out;
        results_out.open("builds/time_results.txt", ios::app);
        if(!results_out.is_open()){
            results_out.open("builds/time_results.txt", ios::trunc);
        }

	       double rmse = calc_rmse(image, "imgs/img_ref_" + to_string(RESOLUTION)+ "_" + to_string(ITERATIONS));
		  cout<<"RMSE: "<<rmse<<endl;

        results_out<<chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << ","<< RESOLUTION<<","<<ITERATIONS<<','<<threads.x<<','<<blocks.x<<','<<rmse<<endl;

        results_out.close();
    

    // Write the result to a file
    ofstream matrix_out;

    
    string imgFilePath = "imgs/img_";
    imgFilePath.append(to_string(RESOLUTION)+"_"+to_string(ITERATIONS));

    cout<<imgFilePath<<endl;
    matrix_out.open(imgFilePath.c_str(), ios::trunc);
    if (!matrix_out.is_open())
    {
        cout << "Unable to open file." << endl;
        return -2;
    }

    for (int row = 0; row < HEIGHT; row++)
    {
        for (int col = 0; col < WIDTH; col++)
        {
            matrix_out << image[row * WIDTH + col];

            if (col < WIDTH - 1)
                matrix_out << ',';
        }
        if (row < HEIGHT - 1)
            matrix_out << endl;
    }
    matrix_out.close();

    delete[] image; 
     hipFree(dev_image); // It's here for coding style, but useless
    return 0;
}

